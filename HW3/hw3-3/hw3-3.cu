
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

const int INF = ((1 << 30) - 1);
const int B = 64;
int v_num, e_num, N;
int* h_dist;

void input(const char* infile) {  
	FILE* file = fopen(infile, "rb");
	fread(&v_num, sizeof(int), 1, file);
	fread(&e_num, sizeof(int), 1, file);
	printf("v=%d\n", v_num);
	N = v_num;
	if (N % B != 0) {
		N = N + (B - N % B);
	}
	
	h_dist = (int*)malloc(N * N * sizeof(int));
	
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			if (i==j && i < v_num)
				h_dist[i*N+j] = 0;
			else
				h_dist[i*N+j] = INF;
		}
	}

	int pair[3];
	for (int i = 0; i < e_num; ++i)
	{
		fread(pair, sizeof(int), 3, file);
		h_dist[pair[0]*N + pair[1]] = pair[2];
	}

	fclose(file);
}

void output(const char* outfile) {   
	FILE* file = fopen(outfile, "w");

	for (int i = 0; i < v_num; ++i) {		
		fwrite(&h_dist[i*N], sizeof(int), v_num, file);
	}

	fclose(file);
}

__global__ void phase1(int* d_dist, int r, int N) {  //解掉bank conflict 
	
	__shared__ int s[B][B];

	int x = threadIdx.x;
	int y = threadIdx.y;

	int i = r * B + y;
	int j = r * B + x;

	s[x][y] = d_dist[i * N + j];
	s[x][y + 32] = d_dist[(i + 32) * N + j];
	s[x + 32][y] = d_dist[i * N + (j + 32)];
	s[x + 32][y + 32] = d_dist[(i + 32) * N + (j + 32)];

#pragma unroll 32
	for (int k = 0; k < B; ++k) {
		__syncthreads();

		s[x][y] = min(s[x][y], s[k][y] + s[x][k]);
		s[x][y + 32] = min(s[x][y + 32], s[k][y + 32] + s[x][k]);
		s[x + 32][y] = min(s[x + 32][y], s[k][y] + s[x + 32][k]);
		s[x + 32][y + 32] = min(s[x + 32][y + 32], s[k][y + 32] + s[x + 32][k]);
	}

	d_dist[i * N + j] = s[x][y];
	d_dist[(i + 32) * N + j] = s[x][y + 32];
	d_dist[i * N + (j + 32)] = s[x + 32][y];
	d_dist[(i + 32) * N + (j + 32)] = s[x + 32][y + 32];
}
__global__ void phase2_1(int* d_dist, int r, int N) {
	
	__shared__ int s1[B][B];
	__shared__ int s2[B][B];

	int x = threadIdx.x;
	int y = threadIdx.y;

	// pivot
	int i = r * B + y;
	int j = r * B + x;

	s1[y][x] = d_dist[i * N + j];
	s1[y + 32][x] = d_dist[(i + 32) * N + j];
	s1[y][x + 32] = d_dist[i * N + (j + 32)];
	s1[y + 32][x + 32] = d_dist[(i + 32) * N + (j + 32)];

	// row
	//i = r * B + y;    
	j = blockIdx.x * B + x;

	s2[y][x] = d_dist[i * N + j];
	s2[y + 32][x] = d_dist[(i + 32) * N + j];
	s2[y][x + 32] = d_dist[i * N + (j + 32)];
	s2[y + 32][x + 32] = d_dist[(i + 32) * N + (j + 32)];

	__syncthreads();

#pragma unroll 32
	for (int k = 0; k < B; ++k) {		
		s2[y][x] = min(s2[y][x], s1[y][k] + s2[k][x]);
		s2[y + 32][x] = min(s2[y + 32][x], s1[y + 32][k] + s2[k][x]);
		s2[y][x + 32] = min(s2[y][x + 32], s1[y][k] + s2[k][x + 32]);
		s2[y + 32][x + 32] = min(s2[y + 32][x + 32], s1[y + 32][k] + s2[k][x + 32]);

	}
	d_dist[i * N + j] = s2[y][x];
	d_dist[(i + 32) * N + j] = s2[y + 32][x];
	d_dist[i * N + (j + 32)] = s2[y][x + 32];
	d_dist[(i + 32) * N + (j + 32)] = s2[y + 32][x + 32];
}

__global__ void phase2_2(int* d_dist, int r, int N) {
	
	__shared__ int s1[B][B];
	__shared__ int s2[B][B];

	int x = threadIdx.x;
	int y = threadIdx.y;

	// pivot
	int i = r * B + y;
	int j = r * B + x;

	s1[y][x] = d_dist[i * N + j];
	s1[y + 32][x] = d_dist[(i + 32) * N + j];
	s1[y][x + 32] = d_dist[i * N + (j + 32)];
	s1[y + 32][x + 32] = d_dist[(i + 32) * N + (j + 32)];

	// col
	i = blockIdx.x * B + y;
	j = r * B + x;

	s2[y][x] = d_dist[i * N + j];
	s2[y + 32][x] = d_dist[(i + 32) * N + j];
	s2[y][x + 32] = d_dist[i * N + (j + 32)];
	s2[y + 32][x + 32] = d_dist[(i + 32) * N + (j + 32)];

	__syncthreads();

#pragma unroll 32
	for (int k = 0; k < B; ++k) {
		s2[y][x] = min(s2[y][x], s2[y][k] + s1[k][x]);
		s2[y + 32][x] = min(s2[y + 32][x], s2[y + 32][k] + s1[k][x]);
		s2[y][x + 32] = min(s2[y][x + 32], s2[y][k] + s1[k][x + 32]);
		s2[y + 32][x + 32] = min(s2[y + 32][x + 32], s2[y + 32][k] + s1[k][x + 32]);

	}

	d_dist[i * N + j] = s2[y][x];
	d_dist[(i + 32) * N + j] = s2[y + 32][x];
	d_dist[i * N + (j + 32)] = s2[y][x + 32];
	d_dist[(i + 32) * N + (j + 32)] = s2[y + 32][x + 32];
}

__global__ void phase3(int* d_dist, int r, int N, int offset) {
	
	__shared__ int s1[B][B];   //self
	__shared__ int s2[B][B];  //row
	__shared__ int s3[B][B];   //col

	int x = threadIdx.x;
	int y = threadIdx.y;

	int i; 
	int j; 

	//row
	i = r * B + y;
	j = blockIdx.x * B + x;

	s2[y][x] = d_dist[i * N + j];
	s2[y + 32][x] = d_dist[(i + 32) * N + j];
	s2[y][x + 32] = d_dist[i * N + (j + 32)];
	s2[y + 32][x + 32] = d_dist[(i + 32) * N + (j + 32)];

	i = (blockIdx.y + offset) * B + y;
	j = r * B + x;

	s3[y][x] = d_dist[i * N + j];
	s3[y + 32][x] = d_dist[(i + 32) * N + j];
	s3[y][x + 32] = d_dist[i * N + (j + 32)];
	s3[y + 32][x + 32] = d_dist[(i + 32) * N + (j + 32)];

	__syncthreads();

	i = (blockIdx.y + offset) * B + y;
	j = blockIdx.x * B + x;

	s1[y][x] = d_dist[i * N + j];
	s1[y + 32][x] = d_dist[(i + 32) * N + j];
	s1[y][x + 32] = d_dist[i * N + (j + 32)];
	s1[y + 32][x + 32] = d_dist[(i + 32) * N + (j + 32)];

#pragma unroll 32
	for (int k = 0; k < B; ++k) {
		s1[y][x] = min(s1[y][x], s3[y][k] + s2[k][x]);
		s1[y + 32][x] = min(s1[y + 32][x], s3[y + 32][k] + s2[k][x]);
		s1[y][x + 32] = min(s1[y][x + 32], s3[y][k] + s2[k][x + 32]);
		s1[y + 32][x + 32] = min(s1[y + 32][x + 32], s3[y + 32][k] + s2[k][x + 32]);
	}

	d_dist[i * N + j] = s1[y][x];
	d_dist[(i + 32) * N + j] = s1[y + 32][x];
	d_dist[i * N + (j + 32)] = s1[y][x + 32];
	d_dist[(i + 32) * N + (j + 32)] = s1[y + 32][x + 32];
}

int main(int argc, char** argv) {

	input(argv[1]);

	int* d_dist[2];
	
	int gridsize = N / B;
	int p = gridsize / 2;	
	

#pragma omp parallel num_threads(2)
	{
		int id = omp_get_thread_num();

		hipSetDevice(id);
		hipStream_t stream[2];
    	hipStreamCreate(&stream[0]);
    	hipStreamCreate(&stream[1]);

		dim3 grid2(gridsize, 1);
		dim3 block(32, 32);
		dim3 grid3;
		int ydim;

		if(gridsize %2 !=0){
			grid3 = dim3(gridsize, p+id);
			ydim = p+id;
		}else{
			grid3 = dim3(gridsize, p);
			ydim = p;
		}
		
		int offset = id * p;

		hipMalloc((void**)&d_dist[id], N * N * sizeof(int));
#pragma omp barrier
		hipMemcpy(&d_dist[id][offset *  N * B], &h_dist[offset *  N * B], ydim *  N * B * sizeof(int), hipMemcpyHostToDevice);

		for (int r = 0; r < gridsize; ++r) {
			
			if (offset <= r && r < offset + ydim)
			{	
				int index = r * N * B;
				if (id == 0)
					hipMemcpy(&(d_dist[1][index]), &(d_dist[0][index]),
							   sizeof(int) * B * N, hipMemcpyDeviceToDevice);
				else
					hipMemcpy(&(d_dist[0][index]), &(d_dist[1][index]),
							   sizeof(int) * B * N, hipMemcpyDeviceToDevice);
			}
#pragma omp barrier
			phase1<<<1, block>>>(d_dist[id], r, N);
			phase2_1<<<grid2, block,0,stream[0]>>>(d_dist[id], r, N);
			phase2_2<<<grid2, block,0,stream[1]>>>(d_dist[id], r, N);
			phase3<<<grid3, block>>>(d_dist[id], r, N, offset);
		}

		hipMemcpy(&h_dist[offset * N * B], &d_dist[id][offset * N * B], ydim * N * B * sizeof(int), hipMemcpyDeviceToHost);

	}
	
	output(argv[2]);

	return 0;
}
